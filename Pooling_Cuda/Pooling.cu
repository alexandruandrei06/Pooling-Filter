#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "util.h"
// Thread block size
#define BLOCK_SIZE 16

__global__ void pooling_cuda(RGB *bitmap, RGB *new_bitmap, char type, int N, int height, int width){
	int row = blockIdx.y * blockDim.y + threadIdx.y; //i
 	int col = blockIdx.x * blockDim.x + threadIdx.x; //j

	int idx_i, idx_j, idx_max, idx_min;
	RGB min, max;
	int idx;
	int k, m;

	if(row >= height || col >= width){
		return;
	}

	if(type == 'M')	{
		idx = row * width + col;
		max.blue = bitmap[idx].blue;
		max.green = bitmap[idx].green;
		max.red = bitmap[idx].red;
		for(k = 0; k < N; k++){
			for(m = 0; m < N; m++){
				//verifica daca este vecin valid
				idx_i = row - N / 2 + k;
				idx_j = col - N / 2 + m;
				idx_max = idx_i * width + idx_j;
				if(!(idx_i < 0 || idx_j < 0 || 
					idx_i >= height || idx_j >= width)){
					if(bitmap[idx_max].blue > max.blue){
						max.blue = bitmap[idx_max].blue;
					}
					if(bitmap[idx_max].green > max.green){
						max.green = bitmap[idx_max].green;
					}
					if(bitmap[idx_max].red > max.red){
						max.red = bitmap[idx_max].red;
					}
				}
			}
		}
		new_bitmap[idx].blue = max.blue;
		new_bitmap[idx].green = max.green;
		new_bitmap[idx].red = max.red;
	} else {
		idx = row * width + col;
		min.blue = bitmap[idx].blue;
		min.green = bitmap[idx].green;
		min.red = bitmap[idx].red;
		for(k = 0; k < N; k++){
			for(m = 0; m < N; m++){
				//verifica daca este vecin valid
				idx_i = row - N / 2 + k;
				idx_j = col - N / 2 + m;
				idx_min = idx_i * width + idx_j;
				if(!(idx_i < 0 || idx_j < 0 || 
					idx_i >= height || idx_j >= width)){
					if(bitmap[idx_min].blue < min.blue)
						min.blue = bitmap[idx_min].blue;
					if(bitmap[idx_min].green < min.green)
						min.green = bitmap[idx_min].green;
					if(bitmap[idx_min].red < min.red)
						min.red = bitmap[idx_min].red;
				}
			}
		}
		new_bitmap[idx].blue = min.blue;
		new_bitmap[idx].green = min.green;
		new_bitmap[idx].red = min.red;
	}
}

// functie care citeste headerele si bitmap-ul pozei
void read_photo(params *param, bmp_photo *photo){
    FILE *input_fd = fopen(param->input_file,"rb");
	if(input_fd == NULL)
		printf("Nu s-a putut deschide fisierul\n");

	//citire header
	fread(photo->header, sizeof(bmp_fileheader), 1, input_fd);

	//citire infoheader
	fread(photo->infoheader, sizeof(bmp_infoheader), 1, input_fd);
	
    hipMallocManaged(&photo->bitmap, photo->infoheader->width * photo->infoheader->height * sizeof(RGB));

    read_bitmap(photo, input_fd);

    fclose(input_fd);
}


int main(int argc, char *argv[]){

    // alocare memorie pentru argumentele de rulare
    params *param = alloc_params();

    // se citesc argumentele programului
	get_args(argc, argv, param);

    bmp_photo *photo = alloc_photo();

	hipMallocManaged(&photo->bitmap, photo->infoheader->width * photo->infoheader->height * sizeof(RGB));

    //citirea pozei
    read_photo(param, photo);

    bmp_photo *new_photo = alloc_photo();
	hipMallocManaged(&new_photo->bitmap, photo->infoheader->width * photo->infoheader->height * sizeof(RGB));
    memcpy(new_photo->header, photo->header, sizeof(bmp_fileheader));
    memcpy(new_photo->infoheader, photo->infoheader, sizeof(bmp_infoheader));

	
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	size_t grid_size_x = photo->infoheader->width / dimBlock.x;
	if (photo->infoheader->width % BLOCK_SIZE) 
		++grid_size_x;
	
	size_t grid_size_y = photo->infoheader->height / dimBlock.y;
	if (photo->infoheader->height % BLOCK_SIZE) 
		++grid_size_y;

    dim3 dimGrid(grid_size_x, grid_size_y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	//Executia kernelul-ului
	pooling_cuda<<<dimGrid, dimBlock>>>(photo->bitmap,
										new_photo->bitmap,
										'M',
										5, 
										photo->infoheader->height,
										photo->infoheader->width);
	
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("Algorithm took %f milliseconds to execute\n", milliseconds);
 
    //scrierea pozei
    write_photo(param, new_photo);

    // free memory
	hipFree(new_photo->bitmap);
	hipFree(photo->bitmap);
    free_params(param);
    free_photo(new_photo);
    free_photo(photo);
    return 0;
}
